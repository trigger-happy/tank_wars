#include "hip/hip_runtime.h"
/*
	This library is free software; you can redistribute it and/or
	modify it under the terms of the GNU Library General Public
	License version 2 as published by the Free Software Foundation.

	This library is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
	Library General Public License for more details.

	You should have received a copy of the GNU Library General Public License
	along with this library; see the file COPYING.LIB.  If not, write to
	the Free Software Foundation, Inc., 51 Franklin Street, Fifth Floor,
	Boston, MA 02110-1301, USA.
*/

#include <algorithm>
#include <cassert>
#include "game_core/physics.h"
#include "util/util.h"

#define NUM_ITERATIONS 30

using namespace Physics;

#define VERLET_BLOCK 64
#define VERLET_GRID MAX_ARRAY_SIZE/VERLET_BLOCK

__host__ vec2_array::vec2_array(){
	std::fill(x, x + MAX_ARRAY_SIZE, 0);
	std::fill(y, y + MAX_ARRAY_SIZE, 0);
}

__host__ __device__ vec2 vec2_array::get_vec2(u32 id){
	vec2 temp;
	temp.x = x[id];
	temp.y = y[id];
	return temp;
}



__host__ physBody::physBody(){
	std::fill(rotation, rotation + MAX_ARRAY_SIZE, 0);
	std::fill(max_vel, max_vel + MAX_ARRAY_SIZE, 0);
	std::fill(can_collide, can_collide + MAX_ARRAY_SIZE, false);
}



__host__ __device__ PhysObject::PhysObject(PhysRunner* p) : m_runner(p){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for creating a physobject from the device
	#elif !defined(__CUDA_ARCH__)
		// host code
		m_objid = m_runner->get_slot();
	#endif
}

__host__ __device__ PhysObject::~PhysObject(){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host code
		m_runner->free_slot(m_objid);
	#endif
}

__host__ __device__ vec2 PhysObject::get_cur_pos(){
	vec2 temp;
	
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		temp.x = m_runner->m_hostbodies.cur_pos.x[m_objid];
		temp.y = m_runner->m_hostbodies.cur_pos.y[m_objid];
	#endif
	
	return temp;
}

__host__ __device__ vec2 PhysObject::get_acceleration(){
	vec2 temp;
	
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		temp.x = m_runner->m_hostbodies.acceleration.x[m_objid];
		temp.y = m_runner->m_hostbodies.acceleration.y[m_objid];
	#endif
	
	return temp;
}

__host__ __device__ f32 PhysObject::get_rotation(){
	f32 rot = 0;
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		rot = m_runner->m_hostbodies.rotation[m_objid];
	#endif
	
	return rot;
}

__host__ __device__ f32 PhysObject::get_max_velocity(){
	f32 mv = 0;
	
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		mv = m_runner->m_hostbodies.rotation[m_objid];
	#endif
	
	return mv;
}

__host__ __device__ bool PhysObject::is_collidable(){
	bool f = false;
	
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		f = m_runner->m_hostbodies.can_collide[m_objid];
	#endif
	
	return f;
}

__host__ __device__ void PhysObject::set_cur_pos(const vec2& pos){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		m_runner->m_hostbodies.cur_pos.x[m_objid] = pos.x;
		m_runner->m_hostbodies.cur_pos.y[m_objid] = pos.y;
		m_runner->m_hostbodies.old_pos.x[m_objid] = pos.x;
		m_runner->m_hostbodies.old_pos.y[m_objid] = pos.y;
		m_runner->update_dev_mem();
	#endif
}

__host__ __device__ void PhysObject::set_acceleration(const vec2& accel){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		m_runner->m_hostbodies.acceleration.x[m_objid] = accel.x;
		m_runner->m_hostbodies.acceleration.y[m_objid] = accel.y;
		m_runner->update_dev_mem();
	#endif
}

__host__ __device__ void PhysObject::set_rotation(f32 r){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		m_runner->m_hostbodies.rotation[m_objid] = r;
		m_runner->update_dev_mem();
	#endif
}

__host__ __device__ void PhysObject::set_max_velocity(f32 mv){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		m_runner->m_hostbodies.max_vel[m_objid] = mv;
		m_runner->update_dev_mem();
	#endif
}

__host__ __device__ void PhysObject::should_collide(bool f){
	#if __CUDA_ARCH__ == 100
		//TODO: code here for device path
	#elif !defined(__CUDA_ARCH__)
		// host path
		m_runner->m_hostbodies.can_collide[m_objid] = f;
		m_runner->update_dev_mem();
	#endif
}



PhysRunner::PhysRunner() 
: m_free_slots(0), m_first_free_slot(0), m_update_dev_mem(false){
	
}

PhysRunner::~PhysRunner(){
	hipFree(m_pdevbodies);
	hipFree(m_pdevshapes);
}

__host__ void PhysRunner::initialize(){
	// initialize the device memory
	hipMalloc(reinterpret_cast<void**>(&m_pdevbodies), sizeof(physBody));
	hipMalloc(reinterpret_cast<void**>(&m_pdevshapes), sizeof(physShape));
	// copy over to the gpu
	copy_to_device();
}

__global__ void update_verlet(f32 dt, physBody* bodies, physShape* shapes){
	u32 idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	vec2 temp = bodies->cur_pos.get_vec2(idx);
	vec2 newpos;
	newpos.x = bodies->cur_pos.x[idx] - bodies->old_pos.x[idx] +
	bodies->acceleration.x[idx] * dt * dt;
	newpos.y = bodies->cur_pos.y[idx] - bodies->old_pos.y[idx] +
	bodies->acceleration.y[idx] * dt * dt;
	
	// don't let the object exceed maximum velocity
	vec2 maxvel;
	maxvel.x = fabsf(dt * bodies->max_vel[idx] * cosf(bodies->rotation[idx] *
	(static_cast<float>(PI)/180)));
	maxvel.y = fabsf(dt * bodies->max_vel[idx] * sinf(bodies->rotation[idx] *
	(static_cast<float>(PI)/180)));
	if(fabsf(newpos.x) > fabsf(maxvel.x)){
		if(bodies->acceleration.x[idx] < 0){
			newpos.x = -maxvel.x;
		}else{
			newpos.x = maxvel.x;
		}
	}
	if(fabsf(newpos.y) > fabsf(maxvel.y)){
		if(bodies->acceleration.y[idx] < 0){
			newpos.y = -maxvel.y;
		}else{
			newpos.y = maxvel.y;
		}
	}
	
	bodies->cur_pos.x[idx] += newpos.x;
	bodies->cur_pos.y[idx] += newpos.y;
	bodies->old_pos.x[idx] = temp.x;
	bodies->old_pos.y[idx] = temp.y;
}

__host__ void PhysRunner::timestep(f32 dt){
	if(m_update_dev_mem){
		copy_to_device();
		m_update_dev_mem = false;
	}
	
	// convert from millisecond to seconds
	dt /= 1000.0f;
	
	update_verlet<<<VERLET_BLOCK, VERLET_GRID>>>(dt, m_pdevbodies,
												 m_pdevshapes);
												 copy_from_device();
}

__host__ void PhysRunner::copy_from_device(){
	hipMemcpy(&m_hostbodies, m_pdevbodies, sizeof(physBody),
			   hipMemcpyDeviceToHost);
			   hipMemcpy(&m_hostshapes, m_pdevshapes, sizeof(physShape),
						  hipMemcpyDeviceToHost);
}

__host__ void PhysRunner::copy_to_device(){
	hipMemcpy(m_pdevbodies, &m_hostbodies, sizeof(physBody),
			   hipMemcpyHostToDevice);
			   hipMemcpy(m_pdevshapes, &m_hostshapes, sizeof(physShape),
						  hipMemcpyHostToDevice);
}

u32 PhysRunner::get_slot(){
	if(!m_free_slots[m_first_free_slot]){
		m_free_slots[m_first_free_slot] = true;
		return m_first_free_slot++;
	}
	find_next_free_slot();
	
	assert(m_first_free_slot < MAX_ARRAY_SIZE);
	
	return m_first_free_slot++;
}

void PhysRunner::free_slot(u32 id){
	m_free_slots[id] = false;
}

void PhysRunner::find_next_free_slot(){
	// keep incremenenting 
	for(u32 i = 0; i < MAX_ARRAY_SIZE; ++i){
		if(!m_free_slots[i]){
			m_first_free_slot = i;
			return;
		}
	}
	m_first_free_slot = MAX_ARRAY_SIZE;
}
